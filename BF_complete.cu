#include <iostream>
#include <fstream>
#include <string>
#include <cstdio>
#include <stdlib.h>
#include "hip/hip_runtime.h"


using namespace std;

#if defined(NDEBUG)
#define CUDA_CHECK(x)	(x)
#else
#define CUDA_CHECK(x)	do {\
		(x); \
		hipError_t e = hipGetLastError(); \
		if (hipSuccess != e) { \
			printf("cuda failure \"%s\" at %s:%d\n", \
			       hipGetErrorString(e), \
			       __FILE__, __LINE__); \
			exit(1); \
		} \
	} while (0)
#endif


__device__ void my_strcpy(char *dest, const char *src) {
	int i = 0;
	do {
		dest[i] = src[i];
	} while (src[++i] != '\0');
}
__device__ int my_strlen(char *string) {
	int cnt = 0;
	while (string[cnt] != '\0') {
		++cnt;
	}
	return cnt;
}

__device__ int my_comp(char* str1, char* str2, int N) {
	int flag = 0;

	for (int i = 0; i<N; i++) {
		if (str1[i] != str2[i]) {
			flag = 1;
			break;
		}
	}

	return flag;
}

__global__ void bruteforce(char* pass, char* alphabet, char* dest, int N, long long int next) { // N = alphabet length 
	extern __shared__ char s_alphabet[];

	char test[100]; // char test = (char*)malloc(sizeof(char)*N);
	int digit[7] = { 0, };
	int passLen = my_strlen(pass);

	for (int i = 0; i<N; i++)
		s_alphabet[i] = alphabet[i];

	digit[6] = blockIdx.x >= N*N*N ? (int)((blockIdx.x / (N*N*N)) % N) : 0;
	digit[5] = blockIdx.x >= N*N ? (int)((blockIdx.x / (N*N)) % N) : 0;
	digit[4] = blockIdx.x >= N ? (int)((blockIdx.x / N) % N) : 0;
	digit[3] = (int)(blockIdx.x % N);
	digit[2] = threadIdx.x;
	digit[1] = 0;
	digit[0] = 0;

	while (digit[1] < N) {
		for (int i = 0; digit[0] < N; digit[0]++, ++i) {
			test[0] = s_alphabet[digit[0]];

			for (int j = 1; j < passLen; j++) {
				test[j] = s_alphabet[digit[j]];
			}
			test[passLen] = '\0';

			if (!my_comp(pass, test, passLen)) {
				my_strcpy(dest, test);
				dest[passLen] = '\0';
				return;
			}
		}
		++digit[1];
		digit[0] = 0;
	}
}

__global__ void bruteforce_write(char* pass, char* alphabet, char* dest, int N, long long unsigned int ExecutionPerThread, long long unsigned total_len) { // N = alphabet length 
	// we don't use shared memory in this function.

	char test[100]; // char test = (char*)malloc(sizeof(char)*N);
	int digit[7] = { 0, };
	int passLen = my_strlen(pass);
	long long unsigned int idx = 0;
	long long unsigned int dummy = 0;

	digit[6] = blockIdx.x >= N*N*N ? (int)((blockIdx.x / (N*N*N)) % N) : 0;
	digit[5] = blockIdx.x >= N*N ? (int)((blockIdx.x / (N*N)) % N) : 0;
	digit[4] = blockIdx.x >= N ? (int)((blockIdx.x / N) % N) : 0;
	digit[3] = (int)(blockIdx.x % N);
	digit[2] = threadIdx.x;
	digit[1] = 0;
	digit[0] = 0;
	
	// ExecutionPerThread = alphabetLen * alphabeltLen * passLen
	while (digit[1] < N) {
		for (int i = 0; digit[0] < N; digit[0]++, ++i) {
			if (blockIdx.x)
				idx = (threadIdx.x * ExecutionPerThread + (i + dummy) * passLen) * blockDim.x * blockIdx.x;
			else
				idx = threadIdx.x * ExecutionPerThread + (i + dummy) * passLen;

			if (idx > total_len) return;

			dest[idx++] = alphabet[digit[0]];

			for (int j = 1; j < passLen; j++) {
				dest[idx++] = alphabet[digit[j]];
			}
		}
		dummy += N ;
		++digit[1];
		digit[0] = 0;
	}
}

void crackPassword(string, int, int);

int main() {
	system("mode con cols=200 lines=250");

	string password;
	string cracked;
	int operation = 0;
	int numOfChars = -1;

	while (operation != 1 && operation != 2) {
		cout << "******* BRUTE FORCE PROGRAM *******" << endl;
		cout << "******* [1]. JUST FIND " << endl;
		cout << "******* [2]. WRITE " << endl;
		cout << "INPUT NUMBER HERE : ";
		cin >> operation;
	}

	if (operation == 2) {
		while (numOfChars <= 0) {
			cout << "INPUT NUMBER OF CHARACTERS(FROM 1 TO 7) : ";
			cin >> numOfChars;
			for (int i = 0; i < numOfChars; i++)
				password += 'a';
		}
	}
	else {
		cout << "Enter the password to crack : ";
		cin >> password;
	}

	crackPassword(password, operation, numOfChars);

	cout << endl;

	return 0;
}

void crackPassword(string pass, int operation, int numOfChars) {
	hipEvent_t start, stop;
	string alphabet;
	string str("");
	char* result;
	char* d_pass;
	char* d_alphabet;
	char* d_dest;
	int alphabetSet = 1;
	int len;
	int cnt = 0;
	int new_cnt = 0;
	int passLen = pass.length();
	float ms = 0;
	char* temp = (char*)malloc(sizeof(char)*pass.length() + 1);
	ofstream ofs("password.txt");
	long long unsigned int total_len;
	bool isFind = false;
	string line("");

	memset(temp, 0, sizeof(char)*pass.length() + 1);
	result = (char*)malloc(sizeof(char)*pass.length() + 1);
	CUDA_CHECK(hipEventCreate(&start));
	CUDA_CHECK(hipEventCreate(&stop));
	CUDA_CHECK(hipMalloc((void**)&d_pass, sizeof(char)*pass.length() + 1));
	CUDA_CHECK(hipMalloc((void**)&d_dest, sizeof(char)*pass.length() + 1));
	CUDA_CHECK(hipMemcpy(d_pass, pass.c_str(), sizeof(char)*pass.length() + 1, hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(d_dest, temp, sizeof(char)*pass.length() + 1, hipMemcpyHostToDevice));

	CUDA_CHECK(hipEventRecord(start)); // start gpu computing with cuda
	while (1) {
		memset(result, 0, pass.length() + 1);
		switch (alphabetSet) {
		case 1: alphabet = "0123456789";
			if (operation == 1)
				cout << endl << endl << "Testing only digits(0123456789) - 10 Characters, please wait";
			else
				cout << endl << endl << "...writing";
			break;
		case 2: alphabet = "abcdefghijklmnopqrstuvwxyz";
			if (operation == 1)
				cout << endl << endl << "Couldn't find the password, increasing the searching level." << endl << endl << "Testing only lowercase characters(abcdefghijklmnopqrstuvwxyz) - 26 Characters, please wait";
			else
				cout << endl << endl << "...writing";
			break;
		case 3: alphabet = "ABCDEFGHIJKLMNOPQRSTUVWXYZ";
			if (operation == 1)
				cout << endl << endl << "Couldn't find the password, increasing the searching level." << endl << endl << "Testing only uppercase characters(ABCDEFGHIJKLMNOPQRSTUVWXYZ) - 26 Characters, please wait";
			else
				cout << endl << endl << "...writing";
			break;
		case 4: alphabet = "0123456789abcdefghijklmnopqrstuvwxyz";
			if (operation == 1)
				cout << endl << endl << "Couldn't find the password, increasing the searching level." << endl << endl << "Testing lowercase characters and numbers(0123456789abcdefghijklmnopqrstuvwxyz) - 36 Characters, please wait";
			else
				cout << endl << endl << "...writing";
			break;
		case 5: alphabet = "0123456789ABCDEFGHIJKLMNOPQRSTUVWXYZ";
			if (operation == 1)
				cout << endl << endl << "Couldn't find the password, increasing the searching level." << endl << endl << "Testing uppercase characters and numbers(0123456789ABCDEFGHIJKLMNOPQRSTUVWXYZ) - 36 Characters, please wait";
			else
				cout << endl << endl << "...writing";
			break;
		case 6: alphabet = "abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ";
			if (operation == 1)
				cout << endl << endl << "Couldn't find the password, increasing the searching level." << endl << endl << "Testing lowercase, uppercase characters(abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ) - 52 Characters, please wait";
			else
				cout << endl << endl << "...writing";
			break;
		case 7: alphabet = "0123456789abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ";
			if (operation == 1)
				cout << endl << endl << "Couldn't find the password, increasing the searching level." << endl << endl << "Testing lowercase, uppercase characters and numbers(0123456789abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ) - 62 Characters, please wait";
			else
				cout << endl << endl << "...writing";
			break;
		}
		len = alphabet.length();
		CUDA_CHECK(hipMalloc((void**)&d_alphabet, sizeof(char)*len + 1));
		CUDA_CHECK(hipMemcpy(d_alphabet, alphabet.c_str(), sizeof(char)*len + 1, hipMemcpyHostToDevice));

		dim3 threadsPerBlock(len, 1);
		dim3 blocksPerGrid((int)std::pow((float)len, pass.length() < 3 ? 1 : (float)(pass.length() - 3)), 1);
		
		switch (operation) {
			case 1: // JUST FIND
				bruteforce<<<blocksPerGrid, threadsPerBlock, sizeof(char) * len >>>(d_pass, d_alphabet, d_dest, len, 0);
				
				CUDA_CHECK(hipMemcpy(result, d_dest, sizeof(char)*pass.length() + 1, hipMemcpyDeviceToHost));

				str = result;
				if (str.compare(pass) == 0) {
					CUDA_CHECK(hipEventRecord(stop));
					hipEventSynchronize(stop);
					cout << endl << "the password : " << result << endl;
					CUDA_CHECK(hipEventElapsedTime(&ms, start, stop));
					cout << "The time duration  passed : " << ms << "ms" << endl << endl;
					isFind = true;
					free(result);
				}			
				
				break;
			case 2: // WRITE
				total_len = (long long unsigned int)(sizeof(char) *  len*len*pass.length() * threadsPerBlock.x * blocksPerGrid.x);
				CUDA_CHECK( hipMalloc((void**)&d_dest, total_len) );
				bruteforce_write<<<blocksPerGrid, threadsPerBlock>>>(d_pass, d_alphabet, d_dest, len, (long long unsigned int)len*len*pass.length(), total_len );
				result = (char*)malloc(total_len);
				CUDA_CHECK(hipMemcpy(result, d_dest, total_len , hipMemcpyDeviceToHost));
				
				// file write
				while(cnt <= total_len) {
					line = "";
					new_cnt = 0;
					while (new_cnt < passLen) {
						if ( (strchr(alphabet.c_str(), result[cnt]) != NULL) && result[cnt] != '\0' ) {
							line += result[cnt];
							new_cnt++;
						}
						cnt++;
						if (cnt >= total_len)
							break;
					}
					ofs << line << endl;
				}	
				cnt = 0;
				new_cnt = 0;
				CUDA_CHECK(hipFree(d_dest));
				free(result);
				break;
		}

		alphabetSet++;
		CUDA_CHECK(hipFree(d_alphabet));

		if (alphabetSet > 7) 
			break;
		if (isFind == true)
			break;
	}

	CUDA_CHECK(hipFree(d_pass));
	if(operation == 1)
		CUDA_CHECK(hipFree(d_dest));
}
